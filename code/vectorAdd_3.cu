/**
 * Suma dos vectores: C = A + B.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define checkError(ans) { asserError((ans), __FILE__, __LINE__); }
inline void asserError(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define TSET(time)  clock_gettime( CLOCK_MONOTONIC, &(time) )
#define TINT(ts,te) { ( (double) 1000.*( (te).tv_sec - (ts).tv_sec ) + ( (te).tv_nsec - (ts).tv_nsec )/(double) 1.e6 ) }

// Numero maximo de threads por bloque
#define MAX_TH_PER_BLOCK 1024

// Tamanho por defecto de los vectores
#define NELDEF 1000

// Numero de threads por bloque por defecto
#define TPBDEF 256

// Numwero de repeticiones
#define NREPDEF 1

// Tipo de datos
typedef float basetype;

/**
 * Codigo host
 */
__host__ void
h_vectorAdd(const basetype *A, const basetype *B, basetype *C, unsigned int numElements)
{
    for (unsigned int i = 0; i < numElements; ++i)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Codigo CUDA
 */
__global__ void
vectorAdd(const basetype *A, const basetype *B, basetype *C, unsigned int numElements)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Funcion main en el host
 * Parametros: nElementos threadsPerBlock nreps
 */
int
main(int argc, char *argv[])
{
    basetype *A=NULL, *B=NULL, *C=NULL;
    unsigned int numElements = 0, tpb = 0, nreps=1;
    size_t size = 0;

    // Valores para la medida de tiempos
    struct timespec tstart, tend;
    double tint;

    // Tamanho de los vectores
    char *end = NULL;
    numElements = (argc > 1) ? strtoul(argv[1], &end, 10):NELDEF;
    // Tamanho de los vectores en bytes
    size = numElements * sizeof(basetype);

    // Numero de threads por bloque
    tpb = (argc > 2) ? atoi(argv[2]):TPBDEF;
    // Comprueba si es superior al máximo
    tpb = (tpb > MAX_TH_PER_BLOCK) ? TPBDEF:tpb;

    // Numero de repeticiones de la suma
    nreps = (argc > 3) ? atoi(argv[3]):NREPDEF;

    // Caracteristicas del Grid
    dim3 threadsPerBlock( tpb );
    // blocksPerGrid = ceil(numElements/threadsPerBlock)
    dim3 blocksPerGrid( (numElements + threadsPerBlock.x - 1) / threadsPerBlock.x );
    printf("Suma de vectores de %u elementos (%u reps), con %u bloques de %u threads\n",
      numElements, nreps, blocksPerGrid.x, threadsPerBlock.x);

    // Reserva memoria unificada
    TSET(tstart);
    checkError(hipMallocManaged(&A, size));
    checkError(hipMallocManaged(&B, size));
    checkError(hipMallocManaged(&C, size));
    TSET(tend);
    tint = TINT(tstart, tend);
    printf("A: Tiempo reserva memoria unificada: %lf ms\n", tint);

    // Inicializa los vectores
    for (unsigned int i = 0; i < numElements; ++i)
    {
        A[i] = rand() / (basetype)RAND_MAX;
        B[i] = rand() / (basetype)RAND_MAX;
    }

    /*
    * Hace la suma en el host
    */
    TSET(tstart);
    for(unsigned int r = 0; r < nreps; ++r)
        h_vectorAdd(A, B, C, numElements);
    TSET(tend);
    tint = TINT(tstart, tend);
    printf("HOST: Tiempo para hacer %u sumas de vectores de tamaño %u: %lf ms\n", nreps, numElements, tint);

    /*
    * Hace la suma en el dispositivo
    */
    TSET(tstart);
    for(unsigned int r = 0; r < nreps; ++r) {
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);
        checkError(hipDeviceSynchronize());
    }
    TSET(tend);
    tint = TINT(tstart, tend);
    printf("C: Tiempo ejecucion kernel GPU: %lf ms\n", tint);

    // Verifica que la suma es correcta
    for (unsigned int i = 0; i < numElements; ++i)
    {
        if (fabs(C[i] - (A[i] + B[i])) > 1e-5)
        {
            fprintf(stderr, "Verificacion de resultados falla en el elemento %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Libera memoria unificada
    checkError(hipFree(A));
    checkError(hipFree(B));
    checkError(hipFree(C));

    return 0;

}
